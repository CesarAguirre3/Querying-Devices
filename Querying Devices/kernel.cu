#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

int main()
{
	int count;
	hipDeviceProp_t prop;

	hipGetDeviceCount(&count);
	for (int i = 0; i < count; i++)
	{
		hipGetDeviceProperties(&prop, i);
		printf("Infrmation of device %d\n", i);
		printf("Name: %s\n", prop.name);
		printf("Clock rate: %d\n", prop.clockRate);
		printf("Multiprocessor count: %d\n", prop.multiProcessorCount);
		printf("Max thread per block: %d\n", prop.maxThreadsPerBlock);
		printf("Max thread dimensions: (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("\n");
	}


    return 0;
}

